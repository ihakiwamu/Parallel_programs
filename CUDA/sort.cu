#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <iostream>
#include <random>
#include <time.h>

#define vec_size 1000000

int main() {
  
  std::random_device rnd;
  clock_t start,end;
  int size;

  std::cin >> size;

  thrust::device_vector<int> host_a(size);
  thrust::device_vector<int> device_vec(size);

  // create num's
  for(int i = 0; i < host_a.size(); i++){
    host_a[i] = rnd();
  }

  start = clock();

  thrust::copy(host_a.begin(), host_a.end(), device_vec.begin()); // Copy host -> device
  thrust::sort(device_vec.begin(), device_vec.end()); // Start sort
  thrust::copy(device_vec.begin(), device_vec.end(), host_a.begin()); // Copy device -> host

  end = clock();

  std::cout << (double)(end - start) / (double)CLOCKS_PER_SEC << "sec.\n";

  return 0;
}
